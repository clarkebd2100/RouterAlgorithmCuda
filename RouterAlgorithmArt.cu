#include "hip/hip_runtime.h"
/******************|********************|*******************|******************|
 *																			   *
 *								ROUTER ALGORITHM							   *
 *																			   *
 *		PREPARED BY: 	Bennett D. Clarke									   *
 *						University of Illinois ( Chicago )					   *
 *						Chicago, Il.  60610									   *
 *		Date:			April 17, 2013									   	   *
 *																			   *
 *******************************************************************************
 *																			   *
 *	OBJECTIVE:	   	To use a data set of routing nodes and to use a parallel   *
 *					processor to run a single source shortest path algorithm   *
 *					to find the first hop node from the chosen source to any   *
 *					of the nodes on the routing topology.  The code is in      *
 *					Cuda.								                       * 							   		   
 *																			   *
 *	METHODOLOGY:	Use Dykstra's algorithm with uniform cost of 1 per router  *
 *					hop ( 1 per edge ).										   *
 ******************************************************************************** 
 *																				*
 *	DESCRIPTION OF VARIABLES													*
 *																				*
 *	edgeArrayParallel[ mVertex + nEdges ]	Large Array with All vertices		*
 *												and their outbound neighbors	*
 *	EdgeAPCount								Index of edgeArrayParallel for both *
 *												terminus of outbound edges and	*
 *												flags for end of linked list	*
 *	EdgeTable[ mVertices ]					Array of pointers; for each vertex	*
 *												the array entry points to a 	*
 *												linked list of outbound edges	*
 *												each of which is a neighbor of	*
 *												the indexed vertex				*
 *	edgeWeightParallel [ iEdge ] 			Weight set to 1000000 until reduced *
 *	leavingEdgeNo							Count of number of outbound edges   *
 *												from a Vertex					*
 *	processingNode[ ]															*				
 *	vertexArrayParallel[ iVertex ]			Each index represents corresponding	*
 *												vertex and it points to the		*
 *												index in the edgeArrayParallel  *
 *												where its first neighbor is 	*
 *												represented						*
 *							
 *******************************************************************************/
 
 
 
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <map>
#include <cstdlib>
//#include <limits>
#include <cstdio>
#include <new>

//	CUDA INCLUDES
#include <hip/hip_runtime.h>

//  CUDA BY EXAMPLE CODE
#include "../common/book.h"


// DECLARATIONS ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

//#define N	10			// DEFINE FOR THREAD BLOCK NUMBER

using std::cout;
using std::endl;

typedef unsigned int vertex;

//CUDA KERNELS

__global__ void ssss_1 ( int N, bool * processingNode, float *  costToNode , 
			float * updatedCost, int * vertexArrayParallel, int * edgeArrayParallel,
			 float * edgeWeightParallel, int * accessedNode, int * parentArray  )
{
	int tid = blockIdx.x;
	int nid;
	if ( tid < N )
	{ 
		if( processingNode[ tid ] )
		{
			processingNode[ tid ] = false;  // Remove node from the exploring group
			
			// TEST IF ORIGINATING VERTEX HAS NEIGHBORS OTHER THAN ITS INBOUND
			// ROUTES FROM THE SOURCE  --  -1 MEANS NO OUTBOUND EDGES
	 		if ( vertexArrayParallel [ tid ]  != -1 ){
				
				// OUTBOUND EDGE TERMINUS IS NID; VAP[ TID ] GIVES THE INDEX IN EAP
				// WHERE THE OUTBOUND EDGE TERMINUS VERTEX ID CAN BE FOUND
				int eAPIndex = vertexArrayParallel[ tid ];
				nid = edgeArrayParallel [ eAPIndex ];			
				
				// THE ARRAY ENTRIES THAT CORRESPOND TO A LINKED LIST OF EDGES 
				// EMANATING FROM THE EXPANDING NODE WILL TERMINATE IN AN ARRAY ENTRY
				// EQUAL TO -1   -- EACH SEPARATE VALID NID REPRESENTS THE TERMINUS
				// OF A NEIGHBOR NODE  
 				while ( -1 != nid  ){
 					++accessedNode[ nid ];
					if ( updatedCost[ nid ] > costToNode[ tid ] + 
														edgeWeightParallel[ eAPIndex ] )
					{
						updatedCost[ nid ] = 
									costToNode [ tid ] + edgeWeightParallel[ eAPIndex ];
						parentArray[ nid ] = tid;
					}
					
					// INCREMENT THE INDEX TO THE EAP BY 1 TO GET NEXT OUTBOUND EDGE
					// TERMINUS - OTHERWISE KNOWN AS NEIGHBOR
 					//++(vertexArrayParallel[ tid ]);
 					++eAPIndex;
 					nid = edgeArrayParallel [ eAPIndex ];
 				} // END WHILE
				
				// FINISHED PROCESSING ALL OUTBOUND EDGES TO NEIGHBORS
				// RETURN DATA TO CPU is coded in cpu section
 				
 			}  // END if != NULL
		}	// END IF PROCEESINGNODE
	}	//	END IF TID < N
	

} // END KERNEL 1

__global__ void ssss_2 ( int N, bool * processingNode, float *  costToNode , 
			float * updatedCost, int * vertexArrayParallel, int * edgeArrayParallel,
			 							float * edgeWeightParallel  )
{
	int tid = blockIdx.x;
	if ( tid < N )
	{ 
 		if( costToNode[ tid ] > updatedCost[ tid ] )
 		{
 			costToNode[ tid ] = updatedCost[ tid ];
 			processingNode[ tid ] = true;
 		} // END UPDATE PATH TO NODE COST
 			else {
 			updatedCost[ tid ] = costToNode[ tid ];
 		}	
	} // END IF tid is valid
} // END KERNEL 2

// VARIABLES
bool undirectedGraphFlag ( false );


//	A LINE OF DATA FROM INPUT FILE CONSISTING OF 2 VERTICES AND THE COST OF A TRAVERSAL
//  FROM VERTEX A TO VERTEX B
struct DataLineIn{

	public:
	vertex vertexANo;
	vertex vertexBNo;
	float weight;
	
	DataLineIn( vertex initVertexANo, vertex initVertexBNo, float initWeight )
		: vertexANo( initVertexANo ), vertexBNo( initVertexBNo ), weight( initWeight ) 
	{}	


};

//	NODE CONSISTING OF A LINE OF DATA ( VERTEX A, VERTEX B, WEIGHT ) AND PTR
struct Node{

	DataLineIn 	data;
	Node *		next;
	
	Node( DataLineIn initData )
		: data ( initData )
	{
		next = NULL;
	}
	
	Node( DataLineIn initData, Node * initNext )
		: data ( initData ), next ( initNext )
	{}	
	
	Node( vertex initVertexANo, vertex initVertexBNo, float initWeight )
		: data( initVertexANo, initVertexBNo, initWeight), next ( NULL )
	{
		std::cout << "Detailed constructor" << std::endl;	
	}
	 
};

struct EdgeQueue{
	
		Node * 		head;
		Node * 		tail;
		
		EdgeQueue( )
		{
			head = NULL;
			tail = NULL;
		}

};

struct TreeNode{
	
	vertex 									vertexNo;
	float									distance;
	std::map < vertex, TreeNode * > 		childMapPtr;
	TreeNode * 								parent;
	
	TreeNode( vertex initVertexNo, float initDistance )
		: vertexNo ( initVertexNo ), distance ( initDistance )
	{
		parent = NULL;
	}	

};

struct Queue{

	TreeNode *				head;
	TreeNode * 				tail;
	
	Queue( )
	{
		head = NULL;
		tail = NULL;
	}
	
};

// 	VECTOR CONSISTING OF DATA IN ENTRIES ( VERTEX A, VERTEX B, WEIGHT )
std::vector < DataLineIn * > DataIn;


// FUNCTIONS


//	PROVIDES USAGE
static void show_usage(std::string fileName)
{
    std::cerr << "Running RouterAlgorithmArt.cu\n"	
    			"Usage: " << fileName << " <option(s)> DATA INPUT FILE  " << "Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-d,--data_file_path \tSpecify the data inputs path"
            << "\t-u,--undirected graph input option"
            << std::endl;
}




int main ( int argc, char ** argv ){

	if ( argc < 3 ) {
        show_usage( argv[ 0 ] );
        return 1;
    }
    std::string dataIn_Name = argv[ 1 ];
    for ( int i = 1; i < argc; ++i ) {
        std::string arg = argv[ i ];
        if ( (   arg == "-h" ) || ( arg == "--help" ) ) {
            show_usage( argv[ 0 ] );
            return 0;
        } else if ( arg == "-u" ){
        	undirectedGraphFlag = true;
        } else if ( (  arg == "-d") || ( arg == "--data_file_path" ) ) {
            if ( i + 1 < argc ) { 
                dataIn_Name = argv[ ++i ];
            } else {
                std::cerr << "--data_file_path option requires one argument." 
                			<< std::endl;
                return 1;
            }  
        }    
	}	// END FOR LOOP
	
	std::cout << "datafilename is: " << dataIn_Name << std::endl;
	unsigned int mV ( 0 );		// TO INPUT NUMBER OF VERTICES
	unsigned int nE ( 0 );		// TO INPUT NUMBER OF EDGES
	int a, b;					// INTEGER REPRESENTATION OF VERTICES
	double c;					// DOUBLE REPRESENTATION OF WEIGHT
	std::string line;
	std::ifstream dataIn;			// STREAM TO INPUT DATAFILE
	
	// PROCESS INPUT FILE INTO VECTOR OF ENTRIES (VERTEXA, VERTEXB, WEIGHT)
	dataIn.open ( dataIn_Name.c_str() );
	if ( dataIn.is_open() )
  	{
    	//	READ FIRST LINE OF DATA WITH SIZE OF GRAPH AND PRINT
    	dataIn >> mV >> nE;
    	
    	// UNDIRECTED GRAPH OPTION
    	if ( undirectedGraphFlag == true ){
    		nE *= 2;
    	}	
    	std::cout << "mVertices: " << mV << ", nEdges: " << nE << std::endl;
    	
    	// PROCESS BODY OF DATA
    	unsigned int dataCount ( 0 );
    	while ( dataCount < nE )
    	{
      		// READ A LINE OF DATA FROM FILE AND PRINT IT
      		dataIn >> a >> b >> c;
      		std::cout << "a: " << a << ", b: " << b << ", c: " << c << std::endl;
      		DataLineIn *ptr = new DataLineIn( a, b, c );
      		
      		// Enqueue new data element ptr to vector
      		DataIn.push_back( ptr );
      		
      		// UNDIRECTED GRAPH OPTION
      		if ( undirectedGraphFlag == true ) {
				cout << "a: " << b << ", b: " << a << ", c: " << c << endl;
				DataLineIn *ptr = new DataLineIn( b, a, c );
			
				// ENQUEUE
				DataIn.push_back( ptr );
				++dataCount;
      		}
      		
      		// INCREMENT FOR NEXT DATA LINE INPUT
      		++dataCount;
    	}
    	dataIn.close();
    	std::cout << "DataIn contains " << DataIn.size() << " records." << std::endl;
  	}
  	//	IF FILE FAILS TO OPEN
	else std::cout << "Unable to open file";
	
	// CREATE CONST UNSIGNED INT TO USE FOR DECLARATION OF ARRAYS
	// DOES NOT WORK HERE
	const unsigned int mVertices ( mV );		// NUMBER OF VERTICES IN GRAPH
	const unsigned int nEdges ( nE );			// NUMBER OF EDGES IN GRAPH
	
	// INITIALIZE 2D DYNAMIC ADJACENCY TABLE TO AN INFINITE ( 1000000 ) VALUE
//	std::cout << "Create adjacency table " << std::endl;
//	float **  adjacency = new float* [ mVertices ];
//	for ( unsigned int i = 0; i < mVertices; ++i ){
//		adjacency[ mVertices ] = new float[ mVertices ];
//	}	
// 	
// 	std::cout << "Initialize Adjacency Table " << std::endl;
// 	for ( unsigned int iCount = 0; iCount < mVertices; iCount++ ){
// 		for ( unsigned int jCount = 0; jCount < mVertices; jCount++ ){
// 			std::cout << "at iter i,j, " << iCount << "," << jCount << "," << std::flush;
// 			//std::cout	<< adjacency[ iCount ][ jCount ] << std::endl;
// 
// 			adjacency[ iCount ][ jCount ] = 1000000.0;
// 		}	
// 	}
	
	// CREATE EDGE TABLE AND INITIALIZE
	std::cout << "Create and initialize Edge Table " << " Size of node* "
				<< sizeof( Node ** ) << std::endl;
	Node ** EdgeTable = new Node * [ mVertices ];
//		std::cout << "Error in creating EdgeTable array! " << std::endl;
	
	// INITIALIZE ARRAY OF NODE POINTERS TO NULL
	// Node * EdgeTable [ mVertices ];
 	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
  		EdgeTable [ iCount ]  = NULL;
  	}	
	
	// LOAD GRAPH DATA INTO ADJACENCY MATRIX 
	std::cout << "Loading data into adjacency table " << std::cout;
	unsigned int kCount = 0;
// 	while( kCount < DataIn.size() ){
// 		adjacency[ DataIn[ kCount ]->vertexANo ][ DataIn[ kCount ]->vertexBNo ] = 
// 														DataIn[ kCount ]->weight;											
// 		++kCount;
// 	} // END WHILE
	
	// PRINT ADJACENCY TABLE
// 	std::cout << "\nAdjacency Matrix:" << std::endl;
// 	std::cout << "  ";
// 	for( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
// 		std::cout << "   v" << iCount;
// 	}
//	std::cout << "Not used with CUDA" << std::endl;
// 	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
// 		std::cout << "v" << iCount;
// 		for( unsigned int jCount = 0; jCount < mVertices; ++jCount ){
// 			if( adjacency[ iCount ][ jCount ] > 999999.9 )
// 				std::cout << "    " << "-" ;
// 			else
// 				std::cout << "    " << adjacency[ iCount ][ jCount ];
// 		}
// 		std::cout << std::endl;
// 	}
	
	kCount = 0;
	while ( kCount < DataIn.size() ){
			
		// EDGE TABLE  -- EACH DATA LINE OF THE INPUT ( AN EDGE ) IS PLACED IN A NODE
		// THE NODE IS THEN INSERTED IN A LINKED LIST THAT IS UNIQUE FOR EACH VERTEX
		// DATA RECORDS NUMBER IS KCOUNT; EDGETABLE ARRAY 1 FOR EACH VERTEX
		// NOTE AUTHORS USE A SINGLE EDGE ARRAY WHERE THE LINKED LIST ELEMENTS ARE
		// HELD AS CONTIGUOUS ARRAY MEMBERS
		DataLineIn dat = *DataIn[ kCount ];
		Node * nPtr = new Node( dat );
		Node * traveller;
		
		// IF THE EDGE TABLE ENTRY HAS NO LINKED LIST - ADD AS FIRST NODE
		if ( !EdgeTable[ DataIn[ kCount ]->vertexANo ] ){
			EdgeTable[ DataIn[ kCount ]->vertexANo ]  = nPtr;
		} else
		
		// ELSE ADD AT END OF LIST
		{
			traveller = EdgeTable[ DataIn[ kCount ]->vertexANo ];
			while ( traveller->next ){
				traveller = traveller->next;
			}
			traveller->next = nPtr;	
		}		// END IF-ELSE BLOCKS	
		++kCount;
	
	} // END WHILE
	
	//	PRINT EDGE TABLE
	std::cout << "Edge Table:" << std::endl;
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		std::cout << iCount << ":  " ;
		Node * traveller;
		traveller = EdgeTable[ iCount ];
		while  ( traveller ){
			std::cout << iCount << "->" << traveller->data.vertexBNo << "   " ;
			traveller = traveller->next;
		}
		std::cout << std::endl;
	}
	
	
	
	//////////////////// CREATE EDGE ADJACENCY ARRAY FOR CUDA /////////////////////
	
	// SIZE OF EXTENDED ADJACENCY TABLE IS SAME AS NUMBER OF DATA RECORDS
	int * vertexArrayParallel = new int [ mVertices ];  // Each index represents its 
												// respectively numbered vertex
												// the value is the EdgeArray index							
												// of the terminus vertex of
												// the first edge in the linked list
												// Represents edge: index->Edarray[value]
	// INITIALIZE THE VERTEXARRAYPARALLEL ENTRIES TO -1 WHICH WILL REPRESENT
	// A VERTEX WITHOUT OUTBOUND EDGES
	for ( unsigned int iVertex = 0; iVertex < mVertices; ++iVertex ){
		vertexArrayParallel [ iVertex ] = -1;
	}
	// PRINT OUT ARRAY VALUES
//	for ( unsigned int iVertex = 0; iVertex < mVertices; ++iVertex ){
//		std::cout << iVertex << ": " << vertexArrayParallel[ iVertex ] << std::endl;
//	}	
														
	// CREATE AN ARRAY THAT WILL PROVIDE ONE ENTRY FOR EVERY OUT EDGE AND EVERY VERTEX
	// INITIALIZE TO -1  -- -1 WILL BE A FLAG TO REPRESENT THE END OF A LINKED LIST 
	// OF OUTBOUND EDGES
	int * edgeArrayP = new int [ nEdges + mVertices ];
	float * edgeWeightParallel = new float [ nEdges + mVertices ];
	
	// INITIALIZE LARGE PARALLEL ARRAY ( edgeArrayP ) WITH -1
	unsigned int iEdge  ( 0 );
	for ( iEdge = 0; iEdge < (nEdges + mVertices); ++iEdge ){
		edgeArrayP[ iEdge ] = -1;
	}
	
	
	unsigned int i ( 0 );
	cout << "After initialization the edgeArrayP:" << endl;
	for( i = 0; i < (nEdges + mVertices); ++i ){
		std::cout << "edgeArrayP[ " << i << " ] : " << edgeArrayP[ i ] 
					<< ", " << std::endl;		
	}	
	
	// INITIALIZE ARRAY OF WEIGHTS TO INFINITE = 1000000
	// NOTE THAT INDICES MUST MATCH THOSE OF THE EDGEARRAYP SO THAT THERE ARE
	// EDGE + VERTICES ENTRIES AND SOME REMAIN 1000000 AS FLAGS
	for ( iEdge = 0; iEdge < ( nEdges + mVertices ); ++iEdge ){
		edgeWeightParallel [ iEdge ] = 1000000;
	}
	cout << "edgeWeightParallel[ each edge ] array as initialized " << endl;
	for( i = 0; i < ( nEdges + mVertices ); ++i ){
		std::cout << "edgeWeightParallel[ " << i << " ] : " << edgeWeightParallel[ i ] 
					<< ", " << std::endl;		
	}
	
	
	// LOAD DATA
	int EdgeAPCount ( 0 );				// Index in EdgeArrayParallel
	std::cout << "Loading data to parallel arrays" << std::endl;
	for ( unsigned int iVertex = 0; iVertex < mVertices; ++iVertex ){
		Node * traveller;
		traveller = EdgeTable[ iVertex ];  // Each table entry is head of linked list
		int leavingEdgeNo ( 0 );			// Initialize number of outbound edges from V
		
		// FOR EACH OUTBOUND EDGE IN A LINKED LIST - TRAVELLER POINTS TO THE EDGE
		// THAT EMANATES FROM VERTEX NUMBERED IVERTEX
		while ( traveller ){
			
			// LEADINGEDGENO == 0 INDICATES THAT THIS IS THE FIRST OUTBOUND EDGE
			// FROM A SPECIFIC EMANATING VERTEX
			if ( ! leavingEdgeNo )
			{	
				// USE POINTER FOR FIRST EDGE IN EACH VERTICE'S LINKED LIST
				std::cout << "iVertex: " << iVertex << std::endl;
				
				// SET POINTER AND ITERATE FOR FIRST ELEMENT IN LINKED LIST
				std::cout << "Head of a list: " << std::flush;
				//vertexArrayParallel [ iVertex ] = &edgeArrayParallel[ EdgeAPCount ];
				vertexArrayParallel [ iVertex ] = EdgeAPCount;
				std::cout << "vertexA<index> to edgeTable<index>: " << "Vertex no: " 							
							<< iVertex 
							<< " maps to " 
							<< vertexArrayParallel [ iVertex ] << " in the large array" 		
							<< std::endl;
				++leavingEdgeNo;
			}
			
			// RECORD B (A->B) VERTEX IN EDGEARRAY FOR ALL NODES
			edgeArrayP[ EdgeAPCount ] = traveller->data.vertexBNo;	
			edgeWeightParallel[ EdgeAPCount ] = traveller->data.weight;
			std::cout << "General node data points to vertex: " 
						<< "edgeArrayP[ " << EdgeAPCount << " ]: " 
						<< edgeArrayP[ EdgeAPCount ] << " weight is " 
						<< edgeWeightParallel[ EdgeAPCount ] << std::endl;
			++EdgeAPCount;
			
			
			// ITERATE ON TRAVELLER IN LINKED LIST
			traveller = traveller->next;
// 			for( unsigned int i = 0; i < EdgeAPCount; ++i ){
// 				std::cout << "index : " << i << " is " << edgeArrayP[ i ] 
// 					<< ", " << std::endl;
// 			}	
		}	// END WHILE LOOP
		// for loop will increment to next vertex
		// PROVIDE A FLAG ( -1 ) TO SEPARATE ELEMENTS BY THEIR INITIATION VERTEX
		// A OF ( A->B )
		cout << "\t\t\tSentinel value: " 	<< "edgeArrayP[ " << EdgeAPCount << " ]: " 
						<< edgeArrayP[ EdgeAPCount ] << std::endl;
		++EdgeAPCount;
	}  // END FOR LOOP
	// CHECK ARRAYS
	for( int i = 0; i < EdgeAPCount; ++i ){
		std::cout << "edgeArrayP[ " << i << " ] : " << edgeArrayP[ i ] 
					<< ", " << std::endl;
	}
	
	for( i = 0; i < ( EdgeAPCount ); ++i ){
		std::cout << "index : " << i << " is " << edgeWeightParallel[ i ] 
					<< ", " << std::endl;		
	}	
	
	
	std::cout << "Declaring device data structures" << std::endl;
	int * dev_vertexArrayParallel;
	int * dev_edgeArrayP;
	float * dev_edgeWeightParallel;
	
 	hipMalloc( ( void** ) &dev_vertexArrayParallel, mVertices * sizeof( int * ) );
 	hipMalloc( ( void** ) &dev_edgeArrayP, (nEdges + mVertices) * sizeof( int ) );
 	hipMalloc( ( void** ) &dev_edgeWeightParallel, 
 										( nEdges + mVertices ) * sizeof( float ) );
	
	////////////////////////////////////////////////////////////////////////////////
	
	// TRANSFORM TO ARRAY OF VERTICES AT HEAD OF LINKED LISTS
	int singleSourceId ( -1 );
	std::cout << "Enter integer identifier of single source node: " << std::flush;

	std::cin >> singleSourceId;
	
	cout << "Received data: " << singleSourceId << endl;

	
	// CREATE A SINGLE SOURCE DATALINE NODE - THIS POINTS TO ITSELF WITH WEIGHT 0
	float sourceWeight ( 0 );
	//DataLineIn * initDataLine = new DataLineIn( singleSourceId, singleSourceId,
	//																 sourceWeight );
	Node * initNode = new Node( singleSourceId, singleSourceId, sourceWeight );
	std::cout << "SS.weight: " << initNode->data.weight << std::endl;
	
	///////////////// DATA STRUCTURES FOR PARALLEL PROCESSING /////////////////////////
	// CREATE A BOOL ARRAY TO HOLD FLAG FOR NODES BEING PROCESSED
	bool processingNode[ mVertices ];
	
	// CREATE A COST ARRAY AS THE COST OF REACHING EACH NODE FROM THE SOURCE
	float costToNode[ mVertices ];
	
	// CREATE A UPDATE COST ARRAY AS THE TEMPORARILY CALCULATED LEAST COST TO
	// EACH NODE FROM THE SOURCE
	
	float updatedCost[ mVertices ];
	
	// INITIALIZE THE ARRAYS
	//	FALSE FOR NOT CURRENTLY EXAMINED
	// 	1000000.00 TO APPROXIMATE INFINITE COST OF AN UNCONNECTED VERTEX
	for ( unsigned int ii = 0; ii < mVertices; ++ii ){
		processingNode [ ii ] = false;
		updatedCost[ ii ] = costToNode[ ii ] = 1000000.00;
	}
	
	for ( unsigned int ii = 0; ii < mVertices; ++ii ){
		cout << "costToNode[ " << ii << " ] : " << costToNode[ ii ] << endl;
	}
	
	// CREATE PARENT ARRAY TO INDICATE PARENT OF ATTACHED NODE// INITIALIZE TO -1
	int parentArray[ mVertices ];
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		parentArray[ iCount ] = -1;
	}	
	
	// CREATE CORRESPONDING ARRAYS FOR GPU AND CUDA
	
 	bool *dev_processingNode;
 	float *dev_costToNode;
 	float *dev_updatedCost;
 	int *dev_parentArray;
 	
// 	// ALLOCATE MEMORY ON GPU
 	hipMalloc( ( void** ) &dev_processingNode, mVertices * sizeof( bool ) );
 	hipMalloc( ( void** ) &dev_costToNode, mVertices * sizeof( float ) );
 	hipMalloc( ( void** ) &dev_updatedCost, mVertices * sizeof( float ) );
 	hipMalloc( ( void** ) &dev_parentArray, mVertices * sizeof( int ) );
	
	std::cout << "parallel processing arrays initialized" << std::endl;
	
	
	//////////////////////////  END COST ARRAYS //////////////////////////////////////
	
	// CREATE A ROOT NODE FOR USE IN THE EXPLORED QUEUE ( element is the source
	// parent is null )
	// TreeNode has elements: vertexNo, distance to source, ptr to map of children,
	// ptr to parent
	TreeNode * root = new TreeNode( singleSourceId, 0.0 );
	root->parent = NULL;
	
	// CREATE A MAP WITH VERTEX NO = KEY; TREE NODE * = VALUE
	std::map< vertex, TreeNode* > * mapPtr = new std::map<vertex, TreeNode* >();
	root->childMapPtr = *mapPtr;
	
	
	// SET EXPLORED NODE QUEUE
	// ANALOG TO GPU SETTING BOOL TO FALSE
	Queue *exploredPtr = new Queue();
	
	// INITIALIZE ARRAY TO HOLD DISTANCE FROM SOURCE TO EACH NODE ID
	// CPU ANALOG OF COSTTONODE
	float distanceToS [ mVertices ];
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		distanceToS[ iCount ] = -1.0;
	}
	
	// ENTER 0 DISTANCE FOR THE SOURCE NODE
	distanceToS[ singleSourceId ] = 0.0;	
	
	// SET ARRAY "PARENT NODE ARRAY' TO HOLD PTR TO PARENT'S TREENODE
	TreeNode * parentNodeArray[ mVertices ];
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		parentNodeArray[ iCount ] = NULL;
	}
	
	// SET PARENTARRAY FOR GPU CALCULATIONS
	parentArray[ singleSourceId ] = singleSourceId;
	
	//INITIALIZE THE EXPLORING QUEUE WITH THE SINGLE SOURCE
	EdgeQueue *exploringPtr = new EdgeQueue();
	exploringPtr->head = initNode;
	std::cout << "adding to exploring queue: " << exploringPtr->head->data.vertexANo
				<< std::endl;	
				
				
//////////////////////////// DIAGNOSTIC STRUCTURES /////////////////////////////////

	int accessedNode[ mVertices ];
	int accessedNodeReply[ mVertices ];
	int *dev_accessedNode;
	for( int iCount = 0; iCount < mVertices; ++iCount ) {
		accessedNode[ iCount ] = accessedNodeReply[ iCount ] = 0;
	}
	
	hipMalloc( ( void** ) &dev_accessedNode, mVertices * sizeof( int ) );
	

						
//////////////////////// GPU INITIALIZATION WITH SINGLE SOURCE /////////////////////

 	processingNode[ singleSourceId ] = true;
 	costToNode[ singleSourceId ] = 0.0;
	updatedCost[ singleSourceId ] = 0.0;	
	
	
////////////////////////  START GPU PROCESSING LOOP ///////////////////////////////////
	
	hipMemcpy( dev_vertexArrayParallel, vertexArrayParallel, 
				mVertices * sizeof( int  ), hipMemcpyHostToDevice );					
	hipMemcpy( dev_edgeArrayP, edgeArrayP, ( mVertices + nEdges ) * sizeof( int ),
											hipMemcpyHostToDevice );
	hipMemcpy( dev_edgeWeightParallel, edgeWeightParallel, 
				( mVertices + nEdges ) * sizeof( float ), hipMemcpyHostToDevice );	

	bool flagContinue ( true );
	while ( flagContinue ){

		// COPY INITIAL DATA TO GPU
		hipMemcpy( dev_processingNode, processingNode, mVertices * sizeof( bool ),
										hipMemcpyHostToDevice );
		hipMemcpy( dev_costToNode, costToNode, mVertices * sizeof( float ),
										hipMemcpyHostToDevice );	
		hipMemcpy( dev_updatedCost, updatedCost, mVertices * sizeof( float ),
										hipMemcpyHostToDevice );
		hipMemcpy( dev_parentArray, parentArray, mVertices * sizeof( int ), 					
														hipMemcpyHostToDevice );										
										
		//	DIAGNOSTIC FUNCTION
		hipMemcpy( dev_accessedNode, accessedNode, 
				mVertices * sizeof( int  ), hipMemcpyHostToDevice );	
	
				
		cout << "Copied data to gpu " << endl;
						

		// EXECUTE KERNEL ON GPU						
		ssss_1<<< mVertices, 1>>> ( mVertices, dev_processingNode, dev_costToNode, 
							dev_updatedCost, dev_vertexArrayParallel, dev_edgeArrayP, 
							dev_edgeWeightParallel, dev_accessedNode, dev_parentArray );	
															
		cout << "Processed kernel1 " << endl;
		

	
		// RETURN PROCESSED DATA FROM GPU
		HANDLE_ERROR( hipMemcpy( updatedCost, dev_updatedCost, 
											mVertices * sizeof( float ),
											hipMemcpyDeviceToHost ) );
											
		cout << "copied updatedCost " << endl;
//		exit ( 0 );
											
		hipMemcpy( processingNode, dev_processingNode, mVertices * sizeof( bool ),
											hipMemcpyDeviceToHost );
		hipMemcpy( parentArray, dev_parentArray, mVertices * sizeof( int ),
											hipMemcpyDeviceToHost );											
											
		// DIAGNOSTIC STRUCTURE
		hipMemcpy( accessedNodeReply, dev_accessedNode, mVertices* sizeof( int ),
											hipMemcpyDeviceToHost );
											
																									
	/////////////////////    END KERNEL 1   ///////////////////////////////////////////	

	cout << "End kernel 1" << endl;
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		cout << "processing [ " << iCount << " ] : " << processingNode[ iCount ] << endl;
	}
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		cout << "updatedCost [ " << iCount << " ] : " << updatedCost[ iCount ] << endl;
	}
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		cout << "costToNode [ " << iCount << " ] : " << costToNode[ iCount ] << endl;
	}
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		cout << "accessedNodeReply [ " << iCount << " ] : " << accessedNodeReply[ iCount ]
				<< endl;
	}			
	
//	exit ( 0 );

	///////////////////// KERNEL 2 ////////////////////////////////////////////////////

	
		// COPY INITIAL DATA TO GPU
		hipMemcpy( dev_processingNode, processingNode, mVertices * sizeof( bool ),
										hipMemcpyHostToDevice );
		hipMemcpy( dev_costToNode, costToNode, mVertices * sizeof( float ),
										hipMemcpyHostToDevice );	
		hipMemcpy( dev_updatedCost, updatedCost, mVertices * sizeof( float ),
										hipMemcpyHostToDevice );	
	
	
		// EXECUTE KERNEL ON GPU						
		ssss_2<<< mVertices, 1>>> ( mVertices, dev_processingNode, dev_costToNode, 
							dev_updatedCost, dev_vertexArrayParallel, dev_edgeArrayP, 
															dev_edgeWeightParallel );
	
		// RETURN PROCESSED DATA FROM GPU
		hipMemcpy( costToNode, dev_costToNode, mVertices * sizeof( float ),
											hipMemcpyDeviceToHost );
		hipMemcpy( updatedCost, dev_updatedCost, mVertices * sizeof( float ),
											hipMemcpyDeviceToHost );
		hipMemcpy( processingNode, dev_processingNode, mVertices * sizeof( bool ),
											hipMemcpyDeviceToHost );
//		hipMemcpy( parentArray, dev_parentArray, mVertices * sizeof( int ),
//											hipMemcpyDeviceToHost );										
											
		cout << "End kernel2" << endl;

	////////////////////////// END KERNEL 2 ////////////////////////////////////////////
	
		cout << "processingNode after kernel2: " << endl;
		for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
			cout << "processing [ " << iCount << " ] : " << processingNode[ iCount ] 
				<< endl;
		}
		for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
			cout << "updatedCost [ " << iCount << " ] : " << updatedCost[ iCount ] 
				<< endl;
		}
		for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
			cout << "costToNode [ " << iCount << " ] : " << costToNode[ iCount ] 
				<< endl;
		}
		for ( unsigned int iCount = 0; iCount < ( mVertices + nEdges ); ++iCount ){
			cout << "edgeWeightParallel [ " << iCount << " ] : " 
				<< edgeWeightParallel[ iCount ] << endl;
		}
		
		for ( unsigned int i = 0; i < mVertices; ++i ){
			if( processingNode[ i ] ) { 
				flagContinue = true; 
				goto repeatLoopTrue; 
			}
		}
		flagContinue = false;
	repeatLoopTrue:
		continue;																		
	}	// END GPU PROCESSING LOOP
	
	// PRINT RESULTS OF GPU PROCESSING
	cout << "Parent array: " << endl;
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		cout <<  "parentArray[ " << iCount << " ] : " << parentArray[ iCount ] << endl;
	}
	
	cout << "Destination Node\tDistance" << endl;
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		cout << "\t " << iCount << ":\t\t\t" << costToNode[ iCount ] << endl;
	}
	
	cout << "Finished distance" << endl;
	
	int firstHopArray[ mVertices ];
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		firstHopArray[ iCount ] = -1;
	}
	
	cout << "initialized firstHop" << endl;	
	int parent ( -1 );
	int intervenor ( -1 );
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
	
		cout << "FirstHop: " << iCount << endl;
		parent = -1;
		intervenor = -1;
		
		// FOR SOURCE NODE
		if( parentArray[ iCount ] == iCount ){
			firstHopArray[ iCount ] = iCount;
			continue;
		} 
		
		// FOR ISOLATED NODE
		if( parentArray[ iCount ] == -1 ){
			cout << iCount << " is isolated from the source node" << endl;
			continue;
		}
		
		// FOR ALL NON-SOURCE VERTICES - BACK TRACK UNTIL SINGLESOURCEID IS REACHED
		cout << "Not source: " << iCount << endl;
		parent = parentArray[ iCount ];
		intervenor = iCount;
		while ( parent != singleSourceId ){
			cout << iCount << " has parent: " << parent << endl;
			intervenor = parent;
			parent = parentArray[ parent ];
		}
		firstHopArray[ iCount ] = intervenor;
		
	}	// END FOR LOOP
	
	cout << "Destination Node\tFirst Hop Router" << endl;
	for( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		if( firstHopArray [ iCount ] == -1 )
		{
			continue;
		}	
		cout << "\t " << iCount << ":\t\t\t" << firstHopArray[ iCount ] << endl;
	}
		
	exit ( 0 );
	
std::cout << exploringPtr->head << std::endl;

std::cout << exploringPtr->head->data.vertexANo << std::endl;
std::cout << exploringPtr->head->data.weight << std::endl;
	
int xCount = 0;
// EXPLORINGPTR->HEAD IS THE NODE THAT IS BEING SEARCHED FOR THE LEAST COST NEXT HOP
while( exploringPtr->head ){
	// FIND CLOSEST NODE
	Node * trav;
	trav = exploringPtr->head;
	Node * followNode;
	Node * preMinNode;
	followNode = exploringPtr->head;	// PREVIOUS NODE ALLOWS LINK BACK
	
	// APPROXIMATE INFINITE DISTANCE ( COST ) WEIGHT AS 1000000.0 
	// FOR EACH NEW SEARCH FOR NEXT SHORTEST THROUGH LINK TO SOURCE
	float minDist ( 1000000.0 );
	Node * minNode; 
	while ( trav ){
		std::cout << "Checking for min with edge: " << trav->data.vertexANo 
					<< "->" << trav->data.vertexBNo << std::endl;
		if( distanceToS[ trav->data.vertexANo ] < 0.0 ){
			// ERROR CHECK
			std::cout << "ERROR - Negative distance" << std::endl;
		} else {
			// CHECK SOURCE-TO-TRAV + TRAV-TO-TEST EDGE FOR MIN TOTAL COST
			 if ( trav->data.weight + distanceToS[trav->data.vertexANo ] < minDist ) {
				minDist = trav->data.weight + distanceToS[ trav->data.vertexANo ];
				std::cout << "New min is Edge : " << trav->data.vertexANo << "->" 
						<< trav->data.vertexBNo << std::endl;
				std::cout << "weight: " << trav->data.weight << std::endl;		
				minNode = trav;
				preMinNode = followNode;
			 }
		 }
		 // INCREMENT ITERATORS SO THAT FOLLOWNODE IS IMMEDIATELY BEHIND trav
		 // TRAV ITERATES THROUGH ALL NODES THAT ARE IN EXPLORING QUEUE
		 // EXPLORING QUEUE = 
		 // NODE THAT ARE BEING CHECKED AS LINKS BUT CHECKS ARE NOT COMPLETE
		 if ( trav != exploringPtr->head )	
		 	followNode = trav;
		trav = trav->next; 
		std::cout << "end while loop: " << trav << std::endl;
	} // END WHILE LOOP TO SEARCH FOR THE NEXT SHORTEST THROUGH LINK TO THE SOURCE
	
	std::cout << "closest uncatalouged route source to node: " << std::flush;
	std::cout	<< minNode->data.vertexANo << "->" << std::flush;
	std::cout 	<< minNode->data.vertexBNo << ", distance: " << std::flush;
	std::cout			<< minDist << "!" << std::endl;	
	
	// REMOVE CLOSEST NODE FROM EXPLORING QUEUE
	if ( minNode == exploringPtr->head ){
		exploringPtr->head = exploringPtr->head->next;
	} else {
		preMinNode->next = minNode->next;
	}
	if ( minNode == exploringPtr->tail ){
		exploringPtr->tail = preMinNode;
	}	
	std::cout << "Deleted from exploring queue edge: " <<  minNode->data.vertexANo 
				<< "->" << minNode->data.vertexBNo << std::endl;
	
	// PURGE ALL INSTANCES OF REMOVED NODE (THEY ARE GREATER DISTANCE )
	trav = exploringPtr->head;
	followNode = exploringPtr->head;
	while( trav ){
		if ( trav->data.vertexBNo == minNode->data.vertexBNo ){
			if( trav == exploringPtr->head ){
				exploringPtr->head = exploringPtr->head->next;
			} else {
				followNode->next = trav->next;			
			}
			if ( trav == exploringPtr->tail ){
				exploringPtr->tail = followNode;
			}		
		}
		// INCREMENT ITERATORS SO THAT FOLLOWNODE IS IMMEDIATELY BEHIND trav
		 if ( trav != exploringPtr->head )	
		 	followNode = trav;
		trav = trav->next;
	}				

	// ADD REMOVED NODE TO THE EXPLORED QUEUE
		// IF EXPLORED NODE IS EMPTY MAKE THE FIRST NODE (SINGLE SOURCE ) ROOT
	if ( !exploredPtr->head ){
		exploredPtr->head = root;
		exploredPtr->tail = root;
		root->childMapPtr[ singleSourceId ] = root;
		std::cout << "Adding to explored queue root: " << exploredPtr->head->vertexNo
				<< std::endl;
				
		// SET ( LEAVE ) PARENTNODE QUEUE TO NULL
		parentNodeArray[ exploredPtr->head->vertexNo ] = root;
	}
		// IF EXPLORED NODE IS NOT EMPTY ADD A NEW NODE
		// ADD NEWEST NODE TO TREE
	else {
		TreeNode * nwNodePtr = 
						new TreeNode( minNode->data.vertexBNo, minNode->data.weight );
		
		// UPDATE PARENTNODEARRAY FOR NEW EXPLORED NODE
		parentNodeArray[ nwNodePtr->vertexNo ] = nwNodePtr;

		// FIND NWNODEPTR'S PARENT NODE FROM PARENTNODEARRAY AND 
		// ADD PARENT PTR TO NEW NODE, THEN ADD CHILD PTR TO PARENT
		// -node of parent
		nwNodePtr->parent = parentNodeArray[ minNode->data.vertexANo ];
		std::cout << "Parent Node is: " << nwNodePtr->parent->vertexNo << std::endl;
		// Parent adds ptr to child (new addition)
		nwNodePtr->parent->childMapPtr[ nwNodePtr->vertexNo ] = nwNodePtr;
		std::cout << "parent: " << nwNodePtr->parent->vertexNo << " has child in node: "
					<< nwNodePtr->parent->childMapPtr[ nwNodePtr->vertexNo ]->vertexNo
					<< std::endl;
		TreeNode * tnTrav = nwNodePtr->parent;			
		while( tnTrav->parent  ){
			tnTrav->parent->childMapPtr[ nwNodePtr->vertexNo ] = tnTrav;
			std::cout << "parent: " << tnTrav->parent->vertexNo 
					<< "has child on route to " << nwNodePtr->vertexNo << " in node: " 
					<< tnTrav->vertexNo << std::endl;
			tnTrav = tnTrav->parent;
		}
		
		// minNode->data.vertexANo as treenode.vertexNo
	} // END IF ELSE BLOCK
	
	// PRINT TREE
	std::cout << "root: " << root->vertexNo << std::endl;
	//int iCount = 0;
	
	// SET DISTANCE OF NODE ADDED TO EXPLORED QUEUE
	distanceToS[ minNode->data.vertexBNo ] = minDist;
	std::cout << "Setting total distance for node: " << minNode->data.vertexBNo
				<< " at " << minDist << std::endl;
				
	// PRINT EXPLORING QUEUE
	std::cout << "Exploring Queue after purge before expansion: ";
	trav = exploringPtr->head;
	if ( trav ){
		while ( trav ){
			std::cout << trav->data.vertexBNo << ", " ;
			trav = trav->next;
		}
	} else {
		std::cout << "empty queue";
	}		
	std::cout << std::endl;				
	
	// EXPAND CHOSEN NODE IN EXPLORING QUEUE AND ADD TO EXPLORING NODE QUEUE	
	// ADD ALL NODES REACHABLE FROM MINNODE (BUT NOT IN EXPLORED QUEUE )
	// TO THE EXPLORING QUEUE
	// minNode->data

	trav = EdgeTable[ minNode->data.vertexBNo ];
	while( trav ){
		if ( distanceToS[ trav->data.vertexBNo ] < 0.0 ){
			std::cout << "have entry(ies) to add" << std::endl;
			if ( !exploringPtr->head ){
				exploringPtr->head = trav;
				exploringPtr->tail = trav;
				std::cout << "Adding to queue" << std::endl;
			} else {
				exploringPtr->tail->next = trav;
				exploringPtr->tail = trav;
			}
			std::cout << "edge: " << trav->data.vertexANo << "->" 
						<< trav->data.vertexBNo << std::endl;			
		}			
		trav = trav->next;		
	}
	// PRINT EXPLORING QUEUE
	std::cout << "Exploring Queue: ";
	trav = exploringPtr->head;
	while ( trav ){
		std::cout << trav->data.vertexANo << "->" << trav->data.vertexBNo << ", " ;
		trav = trav->next;
	}
	std::cout << std::endl;	
	
	// PRINT DISTANCE ARRAY
	std::cout << "\nNode\tDistance From Source" << std::endl;
	for( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		std::cout << iCount << ": \t\t" << distanceToS[ iCount ] << std::endl;
	}	
	
	// PURGE ALL INSTANCES OF REMOVED NODE (THEY ARE GREATER DISTANCE )
	trav = exploringPtr->head;
	followNode = exploringPtr->head;
	while( trav ){
		if ( distanceToS[ trav->data.vertexBNo ] >= 0.0 ){
			if( trav == exploringPtr->head ){
				exploringPtr->head = exploringPtr->head->next;
			} else {
				followNode->next = trav->next;			
			}
			if ( trav == exploringPtr->tail ){
				exploringPtr->tail = followNode;
			}		
		}
		// INCREMENT ITERATORS SO THAT FOLLOWNODE IS IMMEDIATELY BEHIND trav
		 if ( trav != exploringPtr->head )	
		 	followNode = trav;
		trav = trav->next;
	}
++xCount;
} // END WHILE	

	std::cout << "First hop router list: \n" 
				<< "Destination Node	First Hop Node" << std::endl;
	for ( unsigned int iCount = 0; iCount < mVertices; ++iCount ){
		std::cout << "        " << iCount << "                     " 
				<< root->childMapPtr[ iCount ]->vertexNo << std::endl; 	
	
	}
	
	// DELETE POINTERS TO DYNAMIC OBJECTS

	hipFree ( dev_vertexArrayParallel );
	hipFree ( dev_edgeArrayP );
	hipFree ( dev_edgeWeightParallel );
	hipFree ( dev_processingNode );
 	hipFree ( dev_costToNode );
 	hipFree ( dev_updatedCost );
 	hipFree ( dev_parentArray );
 	hipFree ( dev_accessedNode );
 	

 	delete exploringPtr;
 	delete exploredPtr;
 	delete mapPtr;
 	delete root;
 	delete initNode;
 	
 	
 	delete [] 	edgeWeightParallel;
 	delete []	edgeArrayP;
 	delete [] 	vertexArrayParallel;
 	delete [] 	EdgeTable;
	
//	for ( unsigned int i = 0; i < mVertices; ++i ){
//		delete [] adjacency[ i ];
//	}
//	delete [] adjacency;
	
			
	return 0;

}
